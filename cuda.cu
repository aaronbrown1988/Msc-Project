#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "isinglib2.h"

#define DEBUG 0



__global__ void mykernel(spintype *s, float *ran, float * coupling, int n, int dim, float temperature) {
	int r,i;
	float old_energy, new_energy;
	float test;
	r = ran[threadIdx.x] *n*dim;
	
	old_energy = 0;
	for (i=0; i < s[r].n_neigh; i++) {
			old_energy -= coupling[s[r].neigh_couple[i]]* s[s[r].neighbours[i]].s;
	}
	new_energy = 0;
	s[r].s = -s[r].s;
	for (i=0; i < s[r].n_neigh; i++) {
			new_energy -= coupling[s[r].neigh_couple[i]]* s[s[r].neighbours[i]].s;
	}
	
	if (ran[512+threadIdx.x] > exp(-(new_energy -old_energy)/temperature))
		s[r].s = - s[r].s;
}

int main() {
	int n=50, dim=2;
	int i;
	spintype *h_s, *d_s;
	float *h_r, *d_r;
	h_s = setup(1, 50, 2);
	float coup[3] = {-1,-1,-1};
	double coupl[3] = {-1,-1,-1};
	float *d_coupl;
	coupling = coupl;
	h_r = (float*)malloc(1024*sizeof(float));
	hipMalloc(&d_r, 1024*sizeof(float));
	for (i =0; i < 1024; i ++) {
		h_r[i] = (float) rand()/RAND_MAX;
	}
	
	hipMalloc(&d_coupl, 3*sizeof(float));
	
	hipMalloc(&d_s, pow(n,dim)*sizeof(spintype));
	hipMemcpy(d_s, h_s, pow(n,dim)*sizeof(spintype), hipMemcpyHostToDevice);
	hipMemcpy(d_r, h_r, 1024*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_coupl, coup, pow(n,dim)*sizeof(spintype), hipMemcpyHostToDevice);
	mykernel<<<512,512>>>(d_s, d_r, d_coupl, n, dim, 1.0);
	for (i =0; i < 1024; i ++) {
		h_r[i] = (float) rand()/RAND_MAX;
	}
	hipMemcpy(d_r, h_r, 1024*sizeof(float), hipMemcpyHostToDevice);
	mykernel<<<512,512>>>(d_s, d_r, d_coupl, n, dim, 1.0);
	for (i =0; i < 1024; i ++) {
		h_r[i] = (float) rand()/RAND_MAX;
	}
	hipMemcpy(d_r, h_r, pow(n,dim)*sizeof(spintype), hipMemcpyHostToDevice);
	mykernel<<<512,512>>>(d_s, d_r, d_coupl, n, dim, 1.0);
	for (i =0; i < 1024; i ++) {
		h_r[i] = (float) rand()/RAND_MAX;
	}
	hipMemcpy(d_r, h_r, pow(n,dim)*sizeof(spintype), hipMemcpyHostToDevice);
	mykernel<<<512,512>>>(d_s, d_r, d_coupl, n, dim, 1.0);
		hipMemcpy(d_r, h_r, pow(n,dim)*sizeof(spintype), hipMemcpyHostToDevice);
	hipMemcpy(h_s, d_s, pow(n,dim)*sizeof(spintype), hipMemcpyDeviceToHost);
	printf("Got energy %lf\n ", energy_calc(h_s, n, dim, 0));
	hipFree(d_s);
	cleanup(h_s, n, dim);
	return(0);
}

spintype * setup(int type, int n, int dim) {
	spintype *s;
	s = (spintype*)malloc(pow(n,dim)*sizeof(spintype));
	if (s ==NULL) {
		printf("Couldn;t allocate memory\n");
		exit(EXIT_FAILURE);
	}
	if (type == 1) {
		setupSqrSystem(s,n, dim);	
	} else {
		setupTriSystem(s,n,dim);
	}
	initSpins(s,n,dim);
	return(s);
}

void cleanup(spintype * s, int n, int dim) {
	int i;
	for (i = 0; i < pow(n,dim); i ++) {
		free(s[i].neigh_couple);
		free(s[i].neighbours);
	}
	free(s);
}

double sumover(spintype *s, int n, int dim) {
	int i;
	double result;
	result = 0;
	//printf("%d, %d, %lf\n", n, dim, pow(n,dim));
	
	for (i = 0; i < pow(n,dim); i++) {
		result += (double) s[i].s;
	}
	return result;
}


double magorder(spintype *s, int n, int dim) {
	int i;
	double result;
	if (coupling[0] > 0) {
		DEBUGLINE printf("Calling ferro order\n");
		return(sumover(s,n,dim));
	}
	result = 0;
	DEBUGLINE printf("Running Anti order routine\n");
	for (i = 0; i < pow(n,dim); i++) {
		result += (i%2 == 0) ? s[i].s :-s[i].s;
	}
	return result;
}



void setupSqrSystem(spintype *s, int n, int dim) { 
	int i,j,k;
	int curr_spin;
	for (i = 0; i < n; i ++) {
		for(j = 0; j < n; j++) {
			for (k = 0; k < n; k++ ) {
				if ( dim <= 2)
					k = 0;
				curr_spin = ai(i,j,k,n);	
				s[curr_spin].n_neigh = 2*dim;
				s[curr_spin].neighbours = (int*)malloc(sizeof(int)*s[curr_spin].n_neigh);
				s[curr_spin].neigh_couple = (int*)malloc(sizeof(int)*2*dim);
				s[curr_spin].neighbours[0] = (i < (n-1)) ? ai(i+1,j,k,n) : ai(0,j,k,n);
				s[curr_spin].neighbours[1] = ( i == 0) ? ai(n-1,j,k,n) : ai(i-1,j,k,n);
				s[curr_spin].neigh_couple[0] = 0;
				s[curr_spin].neigh_couple[1] = 0;
				if(dim >= 2) {
					s[curr_spin].neighbours[2] = (j < (n-1)) ? ai(i,j+1,k,n) : ai(i,0,k,n);
					s[curr_spin].neighbours[3] = (j == 0) ? ai(i,n-1,k,n) : ai(i,j-1,k,n);
					s[curr_spin].neigh_couple[2] = 1;
					s[curr_spin].neigh_couple[3] = 1;
					
				}
				if(dim >= 3) {
					s[curr_spin].neighbours[4] = (k < (n-1)) ? ai(i,j,k+1,n) : ai(i,j,0,n);
					s[curr_spin].neighbours[5] = (k == 0) ? ai(i,j,n-1,n): ai(i,j,k-1,n);
					s[curr_spin].neigh_couple[4] = 2;
					s[curr_spin].neigh_couple[5] = 2;
				
				}
				if (dim <= 2)
					k = n;
			}
		}
	}
	
}


void setupTriSystem(spintype *s, int n, int dim) { 
	int i,j,k;
	int curr_spin;
	if (dim < 2) {
		printf("Invalid Dimension for Triangular lattice... defaulting to square\n");
		setupSqrSystem(s,n,dim);
		return;
	}
	for (i = 0; i < n; i ++) {
		for(j = 0; j < n; j++) {
			for (k = 0; k < n; k++ ) {
				if ( dim == 2)
					k = 0;
				curr_spin = ai(i,j,k,n);
				/* Initialise Arrays containing Neighbours and coupling info*/
				s[curr_spin].n_neigh = 2*dim + 2;
				s[curr_spin].neighbours = (int*)malloc(sizeof(int)*s[curr_spin].n_neigh);
				s[curr_spin].neigh_couple = (int*)malloc(sizeof(int)*(2*dim + 2));
				/*Neighbours on a line*/
				s[curr_spin].neighbours[0] = (i < (n-1)) ? ai(i+1,j,k,n) : ai(0,j,k,n);
				s[curr_spin].neighbours[1] = ( i == 0) ? ai(n-1,j,k,n) : ai(i-1,j,k,n);
				s[curr_spin].neigh_couple[0] = 0;
				s[curr_spin].neigh_couple[1] = 0;
				if(dim >= 2) {
					/* neighbours in a plane */
					s[curr_spin].neighbours[2] = (j < (n-1)) ? ai(i,j+1,k,n) : ai(i,0,k,n);
					s[curr_spin].neighbours[3] = (j == 0) ? ai(i,n-1,k,n) : ai(i,j-1,k,n);
					s[curr_spin].neigh_couple[2] = 1;
					s[curr_spin].neigh_couple[3] = 1;
					/*Diagonal Neighbours*/
					s[curr_spin].neighbours[4] = (i<(n-1) && j<(n-1)) ? ai(i+1,j+1,k,n) : ai(0,0,k,n);
					s[curr_spin].neighbours[4] = (i>=(n-1) && j<(n-1)) ? ai(0,j+1,k,n) : s[curr_spin].neighbours[4];
					s[curr_spin].neighbours[4] = (i<(n-1) && j >=(n-1)) ? ai(i+1,0,k,n) :s[curr_spin].neighbours[4]; 
					s[curr_spin].neighbours[5] = (j == 0 && i == 0) ? ai(n-1,n-1,k,n) : ai(i-1,j-1,k,n);
					s[curr_spin].neighbours[5] = (j != 0 && i == 0) ? ai(n-1,j-1,k,n) : s[curr_spin].neighbours[5];
					s[curr_spin].neighbours[5] = (j == 0 && i != 0) ? ai(i-1,n-1,k,n) : s[curr_spin].neighbours[5];
					s[curr_spin].neigh_couple[4] = 2;
					s[curr_spin].neigh_couple[5] = 2;
					
				}
				
				
				if(dim >= 3) {
					/* Links between Planes */
					s[curr_spin].neighbours[6] = (k < (n-1)) ? ai(i,j,k+1,n) : ai(i,j,0,n);
					s[curr_spin].neighbours[7] = (k == 0) ? ai(i,j,n-1,n): ai(i,j,k-1,n);
					s[curr_spin].neigh_couple[6] = 3;
					s[curr_spin].neigh_couple[7] = 3;
				
				}
				if (dim <= 2)
					k = n;
			}
		}
	}
	
}




int ai(int i, int j, int k, int n) {
	return (i + j*n + n*n*k);
}




void initSpins(spintype *s, int n, int dim) {
	int r;
	int i,j;
	j = pow(n,dim);
	for (i = 0; i < j; i++) {
		r = rand();
		s[i].s = 0;
		s[i].s = (r <= RAND_MAX/2)? 1:-1;
		if (s[i].s == 0) {
			printf("Error: rand gave: %d\n",r);
			exit(1);
		}
	}
//	printf("Spins are good\n");

	/* trivial check to make sure all spins initialized*/
	for(i = 0; i< j; i ++) {
		if (s[i].s == 0 ) {
			printf("Error: Initalisation failed.\n");
			printf("Error: Spin %d == 0\n", i);
			exit(1);
		}
	}
}


double energy_calc(spintype * s, int n, int dim, double field) {
	int i,l,j;
	double result;
	result = 0;
	j = pow(n,dim);
	for (i=0; i < j; i ++) {
		for (l = 0; l < s[i].n_neigh; l ++) {
			result -= 0.5*coupling[s[i].neigh_couple[l]] * s[i].s * s[s[i].neighbours[l]].s ;
		}
		result += -s[i].s * field;
	}

	result = (double) result / pow(n,dim);
	return result;
}

	
